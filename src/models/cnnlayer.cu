#include "hip/hip_runtime.h"
#include "../lib/cnnlayer.h"

// Constructor
CNNLayer::CNNLayer(int inputHeight, int inputWidth,
         int dstHeight, int dstWidth, 
         int filterHeight, int filterWidth,
         int strideHeight, int strideWidth,
         int paddingHeight, int paddingWidth,
         int numFilters, int numChannels)
    : inputHeight(inputHeight), inputWidth(inputWidth),
      dstHeight(dstHeight), dstWidth(dstWidth), 
      filterHeight(filterHeight), filterWidth(filterWidth),
      strideHeight(strideHeight), strideWidth(strideWidth),
      paddingHeight(paddingHeight), paddingWidth(paddingWidth),
      numFilters(numFilters), numChannels(numChannels) {
    AllocateMemory();
    SetFilters();
}

// Destructor
CNNLayer::~CNNLayer() {
    FreeMemory();
}

// Allocate memory for GPU data
void CNNLayer::AllocateMemory() {
    size_t size_input = inputWidth * inputHeight * numChannels * sizeof(unsigned char);
    hipMalloc(&deviceInput, size_input);

    size_t size_resized = dstHeight * dstWidth * numChannels * sizeof(unsigned char);
    hipMalloc(&deviceResized, size_resized);

    size_t filter_size = filterHeight * filterWidth * numFilters * numChannels * sizeof(float);
    hipMalloc(&deviceFilters, filter_size);

    convHeight = (dstHeight + 2 * paddingHeight - filterHeight) / strideHeight + 1;
    convWidth = (dstWidth + 2 * paddingWidth - filterWidth) / strideWidth + 1;
    size_t conv_size = convWidth * convHeight * numFilters * sizeof(float);
    hipMalloc(&deviceConv, conv_size);

    size_t act_size = convWidth * convHeight * numFilters * sizeof(float);
    hipMalloc(&deviceAct, act_size);

    poolHeight = (convHeight + 2 * paddingHeight - filterHeight) / strideHeight + 1;
    poolWidth = (convWidth + 2 * paddingWidth - filterWidth) / strideWidth + 1;
    size_t pool_size = poolWidth * poolHeight * numFilters * sizeof(float);
    hipMalloc(&devicePool, pool_size);

    int num_threads_col = TILE_WIDTH;
    int num_threads_row = TILE_WIDTH;

    blockSizeconv = dim3(num_threads_col, num_threads_row, numChannels); 
    gridSizeconv = dim3((convWidth + num_threads_col - 1) / num_threads_col, (convHeight + num_threads_row - 1) / num_threads_row, numFilters);
    sharedMemSizeconv = (TILE_WIDTH + filterWidth  - 1) * (TILE_WIDTH + filterHeight  - 1) * numChannels * sizeof(float);

    blockSizeact = dim3(num_threads_col, num_threads_row, 1); 
    gridSizeact = dim3((convWidth + num_threads_col - 1) / num_threads_col, (convHeight + num_threads_row - 1) / num_threads_row, numFilters);

    blockSizepool = dim3(num_threads_col, num_threads_row, 1); 
    gridSizepool = dim3((poolWidth + num_threads_col - 1) / num_threads_col, (poolHeight + num_threads_row - 1) / num_threads_row, numFilters);
    sharedMemSizepool = (TILE_WIDTH + filterWidth  - 1) * (TILE_WIDTH + filterHeight  - 1) * sizeof(float);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err)
                  << " in File " << __FILE__
                  << " in line " << __LINE__
                  << std::endl;
        exit(EXIT_FAILURE);
    }

}

// Free GPU memory
void CNNLayer::FreeMemory() {
    hipFree(deviceInput);
    hipFree(deviceResized);
    hipFree(deviceFilters);
    hipFree(deviceConv);
    hipFree(devicePool);
    hipFree(deviceAct);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err)
                  << " in File " << __FILE__
                  << " in line " << __LINE__
                  << std::endl;
        exit(EXIT_FAILURE);
    }
}

// Forward pass
void CNNLayer::ForwardPass(unsigned char* hostInput) {

    hipMemset(deviceConv, 0, convWidth * convHeight * numFilters * sizeof(float));
    hipMemset(deviceAct, 0, convWidth * convHeight * numFilters * sizeof(float));
    hipMemset(devicePool, 0, poolWidth * poolHeight * numFilters * sizeof(float));


    // Copy from host to device
    hipError_t err = hipMemcpy(deviceInput, hostInput, inputWidth * inputHeight * numChannels * sizeof(unsigned char), hipMemcpyHostToDevice);

    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err)
                << " in File " << __FILE__
                << " in line " << __LINE__
                << std::endl;
        exit(EXIT_FAILURE);
    }

    // Resize the image
    resizeImageGPU();

    LaunchConvolutionKernel();
    LaunchActivationKernel();
    LaunchMaxPoolingKernel();


}

// Implement convolution kernel launch
void CNNLayer::LaunchConvolutionKernel() {

    convolutionKernelSharedMultiple<<<gridSizeconv, blockSizeconv, sharedMemSizeconv>>>(deviceResized, deviceConv, deviceFilters,
                                                                                        dstHeight, dstWidth,
                                                                                        filterHeight, filterWidth,
                                                                                        strideHeight, strideWidth,
                                                                                        paddingHeight, paddingWidth,
                                                                                        numFilters, numChannels);

    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err)
                  << " in File " << __FILE__
                  << " in line " << __LINE__
                  << std::endl;
        exit(EXIT_FAILURE);
    }


    // float* singlefitler = deviceConv;
    // int output_size = convHeight * convWidth;
    // float* hostOutputfloat = new float[output_size]; 
    // unsigned char* hostOutputuchar = new unsigned char[output_size];

    // err = hipMemcpy(hostOutputfloat, singlefitler, output_size * sizeof(float), hipMemcpyDeviceToHost);

    // if (err != hipSuccess) {
    //     std::cerr << "CUDA error: " << hipGetErrorString(err)
    //                 << " in File " << __FILE__
    //                 << " in line " << __LINE__
    //                 << std::endl;
    //     exit(EXIT_FAILURE);
    // }

    // float minRange = *std::min_element(hostOutputfloat, hostOutputfloat + output_size);
    // float maxRange = *std::max_element(hostOutputfloat, hostOutputfloat + output_size);

    // std::cout << minRange << std::endl;
    // std::cout << maxRange << std::endl;

    // if (maxRange == minRange) {
    //     std::fill(hostOutputuchar, hostOutputuchar + output_size, 0);  
    // } else {
    //     for (int i = 0; i < output_size; ++i) {
    //         unsigned char scaledValue = static_cast<unsigned char>(255.0f * (hostOutputfloat[i] - minRange) / (maxRange - minRange));
    //         hostOutputuchar[i] = scaledValue;
    //     }
    // }

    // cv::Mat convMat(convHeight, convWidth, CV_MAKETYPE(CV_8U, 1), hostOutputuchar);
    // cv::imwrite("./output/temp.png", convMat);


    // delete[] hostOutputfloat;
    // delete[] hostOutputuchar;

}

// Implement activation kernel launch
void CNNLayer::LaunchActivationKernel() {

    reluKernelMultiple<<<gridSizeact, blockSizeact>>>(deviceConv, deviceAct, convWidth, convHeight, numFilters);

    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err)
                  << " in File " << __FILE__
                  << " in line " << __LINE__
                  << std::endl;
        exit(EXIT_FAILURE);
    }

}

// Implement max pooling kernel launch
void CNNLayer::LaunchMaxPoolingKernel() {
    MaxPoolingKernelSharedMultiple<<<gridSizepool, blockSizepool, sharedMemSizepool>>>(deviceAct, devicePool,
                                                                                        convHeight, convWidth,
                                                                                        filterHeight, filterWidth,
                                                                                        strideHeight, strideWidth,
                                                                                        paddingHeight, paddingWidth, numFilters);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err)
                  << " in File " << __FILE__
                  << " in line " << __LINE__
                  << std::endl;
        exit(EXIT_FAILURE);
    }

    // float* singlefitler = devicePool + 0 * poolHeight * poolWidth;
    // int output_size = poolWidth * poolHeight;
    // float* hostOutputfloat = new float[output_size]; 
    // unsigned char* hostOutputuchar = new unsigned char[output_size];

    // err = hipMemcpy(hostOutputfloat, singlefitler, output_size * sizeof(float), hipMemcpyDeviceToHost);

    // if (err != hipSuccess) {
    //     std::cerr << "CUDA error: " << hipGetErrorString(err)
    //                 << " in File " << __FILE__
    //                 << " in line " << __LINE__
    //                 << std::endl;
    //     exit(EXIT_FAILURE);
    // }

    // float minRange = *std::min_element(hostOutputfloat, hostOutputfloat + output_size);
    // float maxRange = *std::max_element(hostOutputfloat, hostOutputfloat + output_size);

    // std::cout << minRange << std::endl;
    // std::cout << maxRange << std::endl;

    // if (maxRange == minRange) {
    //     std::fill(hostOutputuchar, hostOutputuchar + output_size, 0);  
    // } else {
    //     for (int i = 0; i < output_size; ++i) {
    //         unsigned char scaledValue = static_cast<unsigned char>(255.0f * (hostOutputfloat[i] - minRange) / (maxRange - minRange));
    //         hostOutputuchar[i] = scaledValue;
    //     }
    // }

    // cv::Mat convMat(poolHeight, poolWidth, CV_MAKETYPE(CV_8U, 1), hostOutputuchar);
    // cv::imwrite("./output/temp1.png", convMat);


    // delete[] hostOutputfloat;
    // delete[] hostOutputuchar;
}


// Set filters from host to device
void CNNLayer::SetFilters() {
    int filter_num_elements = filterHeight * filterWidth * numFilters * numChannels;
    initializeWeights<<<1, filter_num_elements>>>(deviceFilters, filter_num_elements, 1234ULL, -0.5f, 0.5f);
}

void CNNLayer::resizeImageGPU() {

    NppiSize srcSize = {inputWidth, inputHeight}; // Source size
    NppiSize dstSize = {dstWidth, dstHeight}; // Destination size
    NppiRect srcRectROI = {0, 0, inputWidth, inputHeight}; // Source ROI
    NppiRect dstRectROI = {0, 0, dstWidth, dstHeight}; // Destination ROI
    size_t srcStep = inputWidth * numChannels * sizeof(unsigned char); // Row step for source image
    size_t dstStep = dstWidth * numChannels * sizeof(unsigned char); // Row step for destination image
    if (numChannels == 3){

        NppStatus status = nppiResize_8u_C3R(
            deviceInput, srcStep, srcSize, srcRectROI,
            deviceResized, dstStep, dstSize, dstRectROI,
            NPPI_INTER_LINEAR
        );

        if (status != NPP_SUCCESS) {
            std::cerr << "NPP error: " << status << std::endl;
        }

    } else if (numChannels == 4){

        NppStatus status = nppiResize_8u_C4R(
            deviceInput, srcStep, srcSize, srcRectROI,
            deviceResized, dstStep, dstSize, dstRectROI,
            NPPI_INTER_LINEAR
        );
    
        if (status != NPP_SUCCESS) {
            std::cerr << "NPP error: " << status << std::endl;
        }
    } else if (numChannels == 1){

        NppStatus status = nppiResize_8u_C1R(
            deviceInput, srcStep, srcSize, srcRectROI,
            deviceResized, dstStep, dstSize, dstRectROI,
            NPPI_INTER_LINEAR
        );
    
        if (status != NPP_SUCCESS) {
            std::cerr << "NPP error: " << status << std::endl;
        }
    }

    hipDeviceSynchronize();

}


// Get output from device to host
std::tuple<int, int, float*> CNNLayer::GetOutput() {

    float* output = devicePool + 0 * poolHeight * poolWidth;

    return {poolWidth, poolHeight, devicePool};
}
