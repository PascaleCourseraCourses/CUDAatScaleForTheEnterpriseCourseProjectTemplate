#include <../lib/cnnlayer.h>

#include <../lib/utils.h>

#include <random>

std::tuple<std::vector<unsigned char*>, int, int, int, std::vector<std::string>> read_images(const fs::path& directory) {
    std::vector<unsigned char*> images;
    std::vector<std::string> basenames;
    int width = 0;
    int height = 0;
    int channels = 0;

    for (const auto& entry : fs::directory_iterator(directory)) {

        if (entry.is_regular_file() && entry.path().extension() == ".png") {
            // Read the image in grayscale
            cv::Mat img = cv::imread(entry.path().string(), cv::IMREAD_UNCHANGED);
            std::string basename = entry.path().stem().string(); 

            if (!img.empty()) {
                width = img.cols;
                height = img.rows;
                channels = img.channels();
                size_t img_size = width * height * channels * sizeof(unsigned char);
                unsigned char* image = AllocateHostMemory<unsigned char>(img_size, "pinned");
                std::memcpy(image, img.data, img_size);
                images.push_back(image);
                basenames.push_back(basename);

            } else {
                std::cerr << "Failed to load image: " << entry.path() << std::endl;
            }
        } else {
            std::cerr << "Entry is not a regular file or not a PNG: " << entry.path() << std::endl;
        }
    }

    return {images, width, height, channels, basenames}; 
}

std::tuple<std::string, int, int> parseArguments(int argc, char* argv[]) {
    // Initialize default values
    std::string directory = "../data/train/mnist_images";
    int dstWidth = 320;
    int dstHeight = 240;

    // Iterate through command-line arguments
    for (int i = 1; i < argc; ++i) {
        std::string arg = argv[i];

        // Check for the directory flag
        if (arg == "-d" && i + 1 < argc) {
            directory = argv[++i];
        }

        // Check for the width flag
        else if (arg == "-w" && i + 1 < argc) {
            try {
                dstWidth = std::stoi(argv[++i]);
            } catch (const std::invalid_argument& e) {
                std::cerr << "Invalid width value provided. Using default value 320." << std::endl;
            }
        }
        // Check for the height flag
        else if (arg == "-h" && i + 1 < argc) {
            try {
                dstHeight = std::stoi(argv[++i]);
            } catch (const std::invalid_argument& e) {
                std::cerr << "Invalid height value provided. Using default value 240." << std::endl;
            }
        }
    }

    std::cout << "Data Path: " << directory << std::endl;
    std::cout << "Width: " << dstWidth << std::endl;
    std::cout << "Height: " << dstHeight << std::endl;


    return {directory, dstWidth, dstHeight};
}


__host__ void convertToUnsignedChar(const float* input, unsigned char* output, int size) {

    float minRange = *std::min_element(input, input + size);
    float maxRange = *std::max_element(input, input + size);

    // std::cout << minRange << std::endl;
    // std::cout << maxRange << std::endl;

    if (maxRange == minRange) {
        std::fill(output, output + size, 0);  
    } else {
        for (int i = 0; i < size; ++i) {
            unsigned char scaledValue = static_cast<unsigned char>(255.0f * (input[i] - minRange) / (maxRange - minRange));
            output[i] = scaledValue;
        }
    }
}


__host__ void save_image(int outputWidth, int outputHeight, const float* convImage, int numChannels, std::string filename){

    // Calculate size of image
    int output_size = outputWidth * outputHeight * numChannels;
    size_t conv_size = output_size * sizeof(float);

    // Allocate dynamic memory to host image with flot and unsigned char types
    float* h_conv_image = new float[output_size]; // "new" for malloc
    unsigned char* output = new unsigned char[output_size];

    // Copy image to host
    hipError_t err = hipMemcpy(h_conv_image, convImage, conv_size, hipMemcpyDeviceToHost);

    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err)
                  << " in File " << __FILE__
                  << " in line " << __LINE__
                  << std::endl;
        exit(EXIT_FAILURE);
    }

    // Convert float host image to unsigned char host image (0-255)
    convertToUnsignedChar(h_conv_image, output, output_size);

    // Create an OpenCV matrix for host image to use OpenCV functions
    cv::Mat convMat(outputHeight, outputHeight, CV_MAKETYPE(CV_8U, numChannels), output);

    // Save the image
    std::string outputFileName = "./output/output_" + filename + ".png";
    cv::imwrite(outputFileName, convMat);

    delete[] h_conv_image;
    delete[] output;

}


int main(int argc, char* argv[]) {

    auto[directory, dstWidth, dstHeight] = parseArguments(argc, argv);
    
    // Read images
    auto[h_images, srcWidth, srcHeight, numChannels, filenames] = read_images(directory);

    // Initialize convolution paramters
    int filterHeight = 5, filterWidth = 5; 
    int strideHeight = 2, strideWidth = 2;
    int paddingHeight = 2, paddingWidth = 2;
    int numFilters = 1;

    // Construct the network
    CNNLayer SimpleCNN(srcHeight, srcWidth, dstHeight, dstWidth, filterHeight, filterWidth,
                        strideHeight, strideWidth, paddingHeight, paddingWidth, numFilters, numChannels);

    for (size_t i = 0; i < h_images.size(); ++i) {
        const auto& img = h_images[i];
        const auto& filename = filenames[i]; 


        SimpleCNN.ForwardPass(img);

        // Get the output (presumably a convolution output)
        auto[poolWidth, poolHeight, outputimage] = SimpleCNN.GetOutput();

        // Save the result (optionally save with a different name for each image)
        save_image(poolWidth, poolHeight, outputimage, 1, filename);

    }

    return 0;
}
