#include <iostream>
#include <hip/hip_runtime.h>

// CUDA kernel function to add elements of two arrays
__global__ void add(int *a, int *b, int *c, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    int n = 10;
    size_t size = n * sizeof(int);

    // Allocate host memory
    int *h_a = new int[n];
    int *h_b = new int[n];
    int *h_c = new int[n];

    // Initialize host arrays
    for (int i = 0; i < n; ++i) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    // Allocate device memory
    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // Copy data from host to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Launch kernel with 1 block and n threads
    add<<<(n + 255) / 256, 256>>>(d_a, d_b, d_c, n);

    // Copy result from device to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Print result
    std::cout << "Result:\n";
    for (int i = 0; i < n; ++i) {
        std::cout << h_c[i] << " ";
    }
    std::cout << std::endl;

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free host memory
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;

    return 0;
}
